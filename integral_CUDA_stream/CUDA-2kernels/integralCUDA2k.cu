
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <iostream>
#include <cmath>
#include <functional>
#include <ctime>
#include <chrono>




using namespace std;

//function to round the result to two decimal places
double apr(double x){
  double d=round(x*100);
  x=d/100;
  return x;
}

//a function that calculates the integral of a polynomial function of the form f(x)=a*x^n+b*x^(n-1)+...+y*x+z

__global__ void integral2(double x_1, double x_2, double d,double *results){
  
  const int s=2; //polynomial degree
  double w[s+1]={1,1,1}; //array of polynomial coefficients {a,b,c,...}
  double y1=0, y2=0;

  int N=(x_2-x_1)/d; //number of steps
  int t = blockIdx.x * blockDim.x + threadIdx.x; //variable to check if we are not going out of range
  if (t<N)
  {
   for (int i=0; i<s+1; i++){
       y1+=pow(x_1+d*t,i)*w[s-i]; 
       y2+=pow(x_1+d*(t+1),i)*w[s-i]; 
   }
   results[t]=(y1+y2)*0.5*d; //area under a trapezoid with bases y1 and y2 and height d
  }
}
__global__ void integral3(double x_1, double x_2, double d,double *results){
  
  const int s=3; //polynomial degree
  double w[s+1]={1,1,1,1}; //array of polynomial coefficients {a,b,c,...}
  double y1=0, y2=0;

  int N=(x_2-x_1)/d; //number of steps
  int t = blockIdx.x * blockDim.x + threadIdx.x; //variable to check if we are not going out of range
  if (t<N)
  {
   for (int i=0; i<s+1; i++){
       y1+=pow(x_1+d*t,i)*w[s-i]; 
       y2+=pow(x_1+d*(t+1),i)*w[s-i]; 
   }
   results[t]=(y1+y2)*0.5*d; //area under a trapezoid with bases y1 and y2 and height d
  }
}

int main() {
  //measuring the running time of the entire program


  double x_1=-5.,x_2=5.;//integration limits
  double d=0.001; //length of step
  int N=(x_2-x_1)/d; //number of steps

  
  double *result; //array of results
  double r2=0.; //result
  double r3=0.; //result
  size_t size = N*sizeof(double);
  hipMallocManaged(&result, size);

  int threads_per_block = 16;
  int number_of_blocks = (N / threads_per_block) + 1;

  //execution time
  
 std::chrono::steady_clock::time_point s_r1 = std::chrono::steady_clock::now();
    
   integral2<<< number_of_blocks, threads_per_block >>>(x_1, x_2, d, result);
   hipDeviceSynchronize();
   
   for (int i=0; i<N; i++){
       r2+=result[i];
   }
   r2=apr(r2);
   *result=0;
   integral3<<< number_of_blocks, threads_per_block >>>(x_1, x_2, d, result);
   hipDeviceSynchronize();
   
   for (int i=0; i<N; i++){
       r3+=result[i];
   }
   r3=apr(r3);
std::chrono::steady_clock::time_point e_r1 = std::chrono::steady_clock::now();
   
  cout<<"Integral of a second degree polynomial function in limits ("<<x_1<<","<<x_2<<") equals = "<<r2<<endl;
  cout<<"Integral of a third degree polynomial function in limits ("<<x_1<<","<<x_2<<") equals = "<<r3<<endl;
cout<<"Execution time: "<< chrono::duration_cast<chrono::microseconds> (e_r1 - s_r1).count() << " us"<<endl<<endl;

  

  hipFree(result);
}