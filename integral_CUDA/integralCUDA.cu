
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <iostream>
#include <cmath>
#include <functional>
#include <ctime>
#include <chrono>




using namespace std;

//function to round the result to two decimal places
double apr(double x){
  double d=round(x*100);
  x=d/100;
  return x;
}

//a function that calculates the integral of a polynomial function of the form f(x)=a*x^n+b*x^(n-1)+...+y*x+z

__global__ void integral(double x_1, double x_2, double d,double *results){
  
  const int s=2; //polynomial degree
  double w[s+1]={1,1,1}; //array of polynomial coefficients {a,b,c,...}
  double y1=0, y2=0;

  int N=(x_2-x_1)/d; //number of steps
  int t = blockIdx.x * blockDim.x + threadIdx.x; //variable to check if we are not going out of range
  if (t<N)
  {
   for (int i=0; i<s+1; i++){
       y1+=pow(x_1+d*t,i)*w[s-i]; 
       y2+=pow(x_1+d*(t+1),i)*w[s-i]; 
   }
   results[t]=(y1+y2)*0.5*d; //area under a trapezoid with bases y1 and y2 and height d
  }
}

int main() {
  //measuring the running time of the entire program


  double x_1=-5.,x_2=5.;//integration limits
  double d=0.001; //length of step
  int N=(x_2-x_1)/d; //number of steps

  
  double *result; //array of results
  double r=0.; //result
  size_t size = N*sizeof(double);
  hipMallocManaged(&result, size);

  int threads_per_block = 16;
  int number_of_blocks = (N / threads_per_block) + 1;

  //execution time
  
   std::chrono::steady_clock::time_point s_r1 = std::chrono::steady_clock::now();
 
   integral<<< number_of_blocks, threads_per_block >>>(x_1, x_2, d, result);
   hipDeviceSynchronize();
   
   for (int i=0; i<N; i++){
       r+=result[i];
   }
   r=apr(r);
   std::chrono::steady_clock::time_point e_r1 = std::chrono::steady_clock::now();
  
  //result
  

  cout<<"Integral of a polynomial function in limits ("<<x_1<<","<<x_2<<") equals = "<<r<<endl;
  cout<<"Execution time: "<< chrono::duration_cast<chrono::microseconds> (e_r1 - s_r1).count() << " us"<<endl<<endl;
 

  hipFree(result);
}